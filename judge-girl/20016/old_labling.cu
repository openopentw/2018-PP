#include "hip/hip_runtime.h"
#include <stdio.h>

#define BLOCKSIZE 512

__global__ void partialLabeling (const char *cuStr, int *cuPos, int strLen) {
	int begId = blockIdx.x * BLOCKSIZE;
	int endId = (blockIdx.x + 1) * BLOCKSIZE;
	endId = (endId < strLen)? endId : strLen;

	int sum = 0;
	for (int i = begId; i < endId; ++i) {
		if (cuStr[i] == ' ') {
			sum = 0;
		} else {
			++sum;
		}
		cuPos[i] = sum;
	}
}

__global__ void prefixLabeling (const char *cuStr, int *cuPos, int strLen) {
	int begId = (blockIdx.x + 1) * BLOCKSIZE;
	int endId = (blockIdx.x + 2) * BLOCKSIZE;
	endId = (endId < strLen)? endId : strLen;

	if (cuStr[begId - 1] != ' ') {
		int prevPos = cuPos[begId - 1];
		for (int i = begId; i < endId && cuStr[i] != ' '; ++i) {
			cuPos[i] += prevPos;
		}
	}
}

void labeling (const char *cuStr, int *cuPos, int strLen) {
	int block_dim = strLen / BLOCKSIZE;

	partialLabeling <<<block_dim + 1, 1>>> (cuStr, cuPos, strLen);
	hipDeviceSynchronize();

	prefixLabeling <<<block_dim, 1>>> (cuStr, cuPos, strLen);
	hipDeviceSynchronize();
}
