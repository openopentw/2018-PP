
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BLOCKDIM 512

__global__ void partial (const char *cuStr, int *cuPos, int strLen) {
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ int buf[BLOCKDIM];
	if (gid > strLen) {
		return ;
	}

	buf[tid] = (cuStr[gid] == ' ') ? tid : -1;
	for (int i = 1; i <= tid; i <<= 1) {
		__syncthreads();
		int tmp = buf[tid - i];
		__syncthreads();
		buf[tid] = (buf[tid] > tmp)? buf[tid] : tmp;
	}
	__syncthreads();
	cuPos[gid] = tid - buf[tid];
}

__global__ void fix (int *cuPos, int strLen) {
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < strLen && cuPos[gid] == tid + 1) {
		cuPos[gid] += cuPos[gid - tid - 1];
	}
}

void labeling (const char *cuStr, int *cuPos, int strLen) {
	int block_dim = strLen / BLOCKDIM;
	partial <<<block_dim + 1, BLOCKDIM>>> (cuStr, cuPos, strLen);
	fix <<<block_dim, BLOCKDIM>>> (cuPos + BLOCKDIM, strLen - BLOCKDIM);
}
